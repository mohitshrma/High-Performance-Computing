#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) 
{
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

__device__ int is_a_match(char *attempt) {
  char a[] = "FR2138";
  char b[] = "AV7264";
  char c[] = "WW2246";
  char d[] = "YL6075";


  char *first = attempt;
  char *second = attempt;
  char *third = attempt;
  char *fourth = attempt;
  char *password1 = a;
  char *password2 = b;
  char *password3 = c;
  char *password4 = d;

  while(*first == *password1) { 
   if(*first == '\0') 
    {
	printf("Password Found: %s\n",a);
      break;
    }

    first++;
    password1++;
  }
	
  while(*second == *password2) { 
   if(*second == '\0') 
    {
	printf("Password Found: %s\n",b);
      break;
    }

    second++;
    password2++;
  }

  while(*third == *password3) { 
   if(*third == '\0') 
    {
	printf("Password Found: %s\n",c);
      break;
    }

    third++;
    password3++;
  }

  while(*fourth == *password4) { 
   if(*fourth == '\0') 
    {
	printf("Password Found: %s\n",d);
      return 1;
    }

    fourth++;
    password4++;
  }
  return 0;

}


/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char w, x, y, z;
  
  char password[7];                                                 
  password[6] = '\0';

int q = blockIdx.x+65;
int t = threadIdx.x+65;
char firstInitial = q; 
char secondInitial = t; 
    
password[0] = firstInitial;
password[1] = secondInitial;
	for(w='0'; w<='9'; w++){
	  for(x='0'; x<='9'; x++){
	   for(y='0'; y<='9'; y++){
	     for(z='0'; z<='9'; z++){
	        password[2] = w;
	        password[3] = x;
	        password[4] = y;
	        password[5] = z; 
	      if(is_a_match(password)) {
		//printf("Success");
	      } 
             else {
	        //printf("tried: %s\n", password);		  
	         }
	      }
	   }
	}
    }
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed
					, (time_elapsed/1.0e9)); 
  return 0;
}



